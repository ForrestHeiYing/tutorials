#include "hip/hip_runtime.h"
/**
Copyright (c) Facebook, Inc. and its affiliates.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "caffe2/operators/add5_op.h"

#include "caffe2/core/context_gpu.h"
#include "caffe2/utils/math.h"

namespace caffe2 {

template <typename T>
__global__ void Add5Kernel(const int N, const T* data, T* output) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    // TODO - 3
  }
}

template <>
template <typename T>
bool Add5Op<HIPContext>::DoRunWithType() {
  const auto& data = Input(DATA);
  const auto N = data.size();
  const auto* data_ptr = data.template data<T>();
  auto* output = Output(0);
  output->ResizeLike(data);
  auto* output_ptr = output->template mutable_data<T>();

  Add5Kernel<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(N, data_ptr, output_ptr);
  return true;
}

template <typename T>
__global__ void Add5GradientKernel(const int N, const T* data, T* output) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    // GI[0] = GO[0]
    // TODO - 4
  }
}

template <>
template <typename T>
bool Add5GradientOp<HIPContext>::DoRunWithType() {
  const auto& data = Input(DATA);
  const auto N = data.size();
  const auto* data_ptr = data.template data<T>();
  auto* output = Output(0);
  output->ResizeLike(data);
  auto* output_ptr = output->template mutable_data<T>();

  Add5GradientKernel<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(N, data_ptr, output_ptr);
  return true;
}

REGISTER_HIP_OPERATOR(Add5, Add5Op<HIPContext>);
REGISTER_HIP_OPERATOR(Add5Gradient, Add5GradientOp<HIPContext>);

} // namespace caffe2
